#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/recurrent_network_op.h"

namespace caffe2 {

namespace detail {

template <typename T, typename Context>
void initializeRecurrentInput(
    const RecurrentInput& rc,
    int32_t seqLen,
    int32_t batchSize,
    Workspace* ws,
    Context* context);

namespace {

template <typename T>
__global__
void initRecurrentInput_kernel(
    size_t stateSize,
    const T* input,
    T* state) {
  // index into appropriate target buffer
  const int block_id = blockIdx.x;
  T* state_local = state + block_id*stateSize;

  // copy
  for (int idx=threadIdx.x; idx < stateSize; idx+=blockDim.x) {
    state_local[idx] = input[idx];
  }
}


}; // namespace

template <>
void initializeRecurrentInput<float,HIPContext>(
    const RecurrentInput& rc,
    int32_t seqLen,
    int32_t batchSize,
    Workspace* ws,
    HIPContext* context) {
  auto stateBlob = ws->GetBlob(rc.state);
  CAFFE_ENFORCE(stateBlob);
  auto* state = stateBlob->GetMutable<Tensor<HIPContext>>();

  auto inputBlob = ws->GetBlob(rc.input);
  CAFFE_ENFORCE(inputBlob);
  const auto& input = inputBlob->Get<Tensor<HIPContext>>();
  CAFFE_ENFORCE_GE(input.ndim(), 1, rc.input);
  CAFFE_ENFORCE_LE(input.ndim(), 3, rc.input);

  const auto stateSize = input.dim(input.ndim() - 1);
  // States at [0, ..., T] (inclusive)
  state->Resize(seqLen + 1, batchSize, stateSize);

  if (input.ndim() == 3) {
    CAFFE_ENFORCE_EQ(input.dim(0), 1, rc.input);
  }
  if (input.ndim() >= 2) {
    CAFFE_ENFORCE_EQ(input.dim(input.ndim() - 2), batchSize, rc.input);
    context->Copy<float, HIPContext, HIPContext>(
        batchSize * stateSize,
        input.data<float>(),
        state->mutable_data<float>());
  } else {
    initRecurrentInput_kernel<float><<<batchSize, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
        stateSize,
        input.data<float>(),
        state->mutable_data<float>());
  }
}
template <>
void initializeRecurrentInput<float16,HIPContext>(
    const RecurrentInput& rc,
    int32_t seqLen,
    int32_t batchSize,
    Workspace* ws,
    HIPContext* context) {
  auto stateBlob = ws->GetBlob(rc.state);
  CAFFE_ENFORCE(stateBlob);
  auto* state = stateBlob->GetMutable<Tensor<HIPContext>>();

  auto inputBlob = ws->GetBlob(rc.input);
  CAFFE_ENFORCE(inputBlob);
  const auto& input = inputBlob->Get<Tensor<HIPContext>>();
  CAFFE_ENFORCE_GE(input.ndim(), 1, rc.input);
  CAFFE_ENFORCE_LE(input.ndim(), 3, rc.input);

  const auto stateSize = input.dim(input.ndim() - 1);
  // States at [0, ..., T] (inclusive)
  state->Resize(seqLen + 1, batchSize, stateSize);

  if (input.ndim() == 3) {
    CAFFE_ENFORCE_EQ(input.dim(0), 1, rc.input);
  }
  if (input.ndim() >= 2) {
    CAFFE_ENFORCE_EQ(input.dim(input.ndim() - 2), batchSize, rc.input);
    context->Copy<float16, HIPContext, HIPContext>(
        batchSize * stateSize,
        input.data<float16>(),
        state->mutable_data<float16>());
  } else {
    initRecurrentInput_kernel<float16><<<batchSize, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
        stateSize,
        input.data<float16>(),
        state->mutable_data<float16>());
  }
}

}; // namespace detail

template <>
bool RecurrentNetworkOp<HIPContext>::RunOnDevice() {
  if (Input(0).IsType<float>()) {
    return DoRunWithType<float>();
  } else if (Input(0).IsType<float16>()) {
    return DoRunWithType<float16>();
  } else {
    LOG(FATAL) << "Only float (32bit) and float16 inputs "
               << "are supported by RecurrentNetworkOp, "
               << "but input " << debug_def().input(0) << " has ["
               << Input(0).meta().name() << "] ";
  }
  return false;
}

template <>
bool RecurrentNetworkGradientOp<HIPContext>::RunOnDevice() {
  if (Input(0).IsType<float>()) {
    return DoRunWithType<float>();
  } else if (Input(0).IsType<float16>()) {
    return DoRunWithType<float16>();
  } else {
    LOG(FATAL) << "Only float (32bit) and float16 inputs "
               << "are supported by RecurrentNetworkGradientOp, "
               << "but input " << debug_def().input(0) << " has ["
               << Input(0).meta().name() << "] ";
  }
  return false;
}

template <>
bool AccumulateInputGradientOp<HIPContext>::RunOnDevice() {
  if (Input(1).IsType<float>()) {
    return DoRunWithType<float>();
  } else if (Input(1).IsType<float16>()) {
    return DoRunWithType<float16>();
  } else {
    LOG(FATAL) << "Only float (32bit) and float16 inputs "
               << "are supported by AccumulateInputGradientOp, "
               << "but input " << debug_def().input(1) << " has ["
               << Input(1).meta().name() << "] ";
  }
  return false;
}

REGISTER_HIP_OPERATOR(
    RecurrentNetwork,
    RecurrentNetworkOp<HIPContext>);
REGISTER_HIP_OPERATOR(
    RecurrentNetworkGradient,
    RecurrentNetworkGradientOp<HIPContext>);
REGISTER_HIP_OPERATOR(
    rnn_internal_accumulate_gradient_input,
    AccumulateInputGradientOp<HIPContext>);
REGISTER_HIP_OPERATOR(
    rnn_internal_apply_link,
    RNNApplyLinkOp<float, HIPContext>);
}


} // namespace caffe2
